#include "utils-cuda.h"
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <mutex>
#include <string>
#include <iostream>

unsigned char h_sbox[256] = {
    0x63, 0x7C, 0x77, 0x7B, 0xF2, 0x6B, 0x6F, 0xC5, 0x30, 0x01, 0x67, 0x2B, 0xFE, 0xD7, 0xAB, 0x76,
    0xCA, 0x82, 0xC9, 0x7D, 0xFA, 0x59, 0x47, 0xF0, 0xAD, 0xD4, 0xA2, 0xAF, 0x9C, 0xA4, 0x72, 0xC0,
    0xB7, 0xFD, 0x93, 0x26, 0x36, 0x3F, 0xF7, 0xCC, 0x34, 0xA5, 0xE5, 0xF1, 0x71, 0xD8, 0x31, 0x15,
    0x04, 0xC7, 0x23, 0xC3, 0x18, 0x96, 0x05, 0x9A, 0x07, 0x12, 0x80, 0xE2, 0xEB, 0x27, 0xB2, 0x75,
    0x09, 0x83, 0x2C, 0x1A, 0x1B, 0x6E, 0x5A, 0xA0, 0x52, 0x3B, 0xD6, 0xB3, 0x29, 0xE3, 0x2F, 0x84,
    0x53, 0xD1, 0x00, 0xED, 0x20, 0xFC, 0xB1, 0x5B, 0x6A, 0xCB, 0xBE, 0x39, 0x4A, 0x4C, 0x58, 0xCF,
    0xD0, 0xEF, 0xAA, 0xFB, 0x43, 0x4D, 0x33, 0x85, 0x45, 0xF9, 0x02, 0x7F, 0x50, 0x3C, 0x9F, 0xA8,
    0x51, 0xA3, 0x40, 0x8F, 0x92, 0x9D, 0x38, 0xF5, 0xBC, 0xB6, 0xDA, 0x21, 0x10, 0xFF, 0xF3, 0xD2,
    0xCD, 0x0C, 0x13, 0xEC, 0x5F, 0x97, 0x44, 0x17, 0xC4, 0xA7, 0x7E, 0x3D, 0x64, 0x5D, 0x19, 0x73,
    0x60, 0x81, 0x4F, 0xDC, 0x22, 0x2A, 0x90, 0x88, 0x46, 0xEE, 0xB8, 0x14, 0xDE, 0x5E, 0x0B, 0xDB,
    0xE0, 0x32, 0x3A, 0x0A, 0x49, 0x06, 0x24, 0x5C, 0xC2, 0xD3, 0xAC, 0x62, 0x91, 0x95, 0xE4, 0x79,
    0xE7, 0xC8, 0x37, 0x6D, 0x8D, 0xD5, 0x4E, 0xA9, 0x6C, 0x56, 0xF4, 0xEA, 0x65, 0x7A, 0xAE, 0x08,
    0xBA, 0x78, 0x25, 0x2E, 0x1C, 0xA6, 0xB4, 0xC6, 0xE8, 0xDD, 0x74, 0x1F, 0x4B, 0xBD, 0x8B, 0x8A,
    0x70, 0x3E, 0xB5, 0x66, 0x48, 0x03, 0xF6, 0x0E, 0x61, 0x35, 0x57, 0xB9, 0x86, 0xC1, 0x1D, 0x9E,
    0xE1, 0xF8, 0x98, 0x11, 0x69, 0xD9, 0x8E, 0x94, 0x9B, 0x1E, 0x87, 0xE9, 0xCE, 0x55, 0x28, 0xDF,
    0x8C, 0xA1, 0x89, 0x0D, 0xBF, 0xE6, 0x42, 0x68, 0x41, 0x99, 0x2D, 0x0F, 0xB0, 0x54, 0xBB, 0x16
};

unsigned char h_rcon[11] = {
    0x00, // not used
    0x01, 0x02, 0x04, 0x08, 0x10, 
    0x20, 0x40, 0x80, 0x1B, 0x36
};

// Function to read key or IV from a file
void read_key_or_iv(unsigned char *data, size_t size, const char *filename) {
    FILE *file = fopen(filename, "r");
    if (file == NULL) {
        fprintf(stderr, "Cannot open file: %s\n", filename);
        exit(1);
    }
    for (size_t i = 0; i < size; i++) {
        char buffer[3];
        if (fread(buffer, 1, 2, file) != 2) {
            fprintf(stderr, "Cannot read value from file: %s\n", filename);
            exit(1);
        }
        buffer[2] = '\0'; // Null-terminate the buffer
        data[i] = (unsigned char)strtol(buffer, NULL, 16); // Convert the buffer to a hexadecimal value
    }
    fclose(file);
}

void read_file_as_binary(unsigned char **data, size_t *size, const char *filename) {
    FILE *file = fopen(filename, "rb");
    if (file == NULL) {
        fprintf(stderr, "Cannot open file: %s\n", filename);
        exit(1);
    }

    // Determine the file size
    fseek(file, 0, SEEK_END);
    *size = ftell(file);
    fseek(file, 0, SEEK_SET);

    // Allocate the buffer
    *data = new unsigned char[*size];

    size_t bytesRead = fread(*data, 1, *size, file);
    if (bytesRead != *size) {
        fprintf(stderr, "Failed to read the entire file: %s\n", filename);
        exit(1);
    }

    fclose(file);
}

// Add pinned memory allocation
void read_file_as_binary_v2(unsigned char **data, size_t *size, const char *filename) {
    FILE *file = fopen(filename, "rb");
    if (file == NULL) {
        fprintf(stderr, "Cannot open file: %s\n", filename);
        exit(1);
    }

    // Determine the file size
    fseek(file, 0, SEEK_END);
    *size = ftell(file);
    fseek(file, 0, SEEK_SET);

    // Allocate pinned memory for data
    hipHostMalloc((void**)data, *size * sizeof(unsigned char));

    // Read the file into data
    if (fread(*data, 1, *size, file) != *size) {
        fprintf(stderr, "Cannot read file: %s\n", filename);
        exit(1);
    }

    fclose(file);
}

size_t preprocess(const char *filename, size_t chunkSize, unsigned char ***chunks, size_t **chunkSizes) {
    // Read the file into a buffer
    unsigned char *buffer;
    size_t bufferSize;
    read_file_as_binary_v2(&buffer, &bufferSize, filename);

    // Calculate the number of chunks
    size_t numChunks = (bufferSize + chunkSize - 1) / chunkSize;

    // Allocate pinned memory for the chunks and their sizes
    hipHostMalloc((void***)chunks, numChunks * sizeof(unsigned char*));
    hipHostMalloc((void**)chunkSizes, numChunks * sizeof(size_t));

    // Split the buffer into chunks
    for (size_t i = 0; i < numChunks; i++) {
        // Calculate the size of the current chunk
        size_t currentChunkSize = (i < numChunks - 1) ? chunkSize : (bufferSize % chunkSize);

        // Allocate pinned memory for the current chunk
        hipHostMalloc((void**)&(*chunks)[i], currentChunkSize * sizeof(unsigned char));

        printf("Chunk %zu address: %p\n", i, (*chunks)[i]);  // Print the address of the current chunk

        // Copy the data from the buffer to the current chunk
        memcpy((*chunks)[i], buffer + i * chunkSize, currentChunkSize);

        // Save the size of the current chunk
        (*chunkSizes)[i] = currentChunkSize;
    }

    // Free the buffer
    hipHostFree(buffer);

    return numChunks;
}

void write_encrypted(const unsigned char *ciphertext, size_t size, const char *filename) {
    FILE *file = fopen(filename, "wb");
    if (file == NULL) {
        fprintf(stderr, "Cannot open file: %s\n", filename);
        exit(1);
    }
    if (fwrite(ciphertext, 1, size, file) != size) {
        fprintf(stderr, "Error writing to file: %s\n", filename);
        exit(1);
    }
    fclose(file);
}

void write_encrypted_v2(unsigned char* ciphertext, size_t size, const char* filename) {
    FILE* file = fopen(filename, "ab");
    if (file == NULL) {
        printf("Error opening file: %s\n", filename);
        return;
    }
    if (fwrite(ciphertext, 1, size, file) != size) {
        fprintf(stderr, "Error writing to file: %s\n", filename);
        exit(1);
    }
    fclose(file);
}

std::mutex fileMutex;
void write_encrypted_multithreading(const unsigned char *ciphertext, size_t size, const char *filename) {
    std::lock_guard<std::mutex> lock(fileMutex);

    // Open the file in append mode
    FILE *file = fopen(filename, "ab");
    if (file == NULL) {
        fprintf(stderr, "Cannot open file: %s\n", filename);
        exit(1);
    }

    // Write the data to the file
    size_t written = fwrite(ciphertext, 1, size, file);
    if (written != size) {
        fprintf(stderr, "Failed to write to file: %s\n", filename);
        exit(1);
    }

    fclose(file);
}

std::string getFileExtension(const std::string& filename) {
    size_t pos = filename.rfind('.');
    return (pos == std::string::npos) ? "" : filename.substr(pos);
}

void appendFileExtension(const std::string& filename, const std::string& extension) {
    FILE* file = fopen(filename.c_str(), "ab");
    if (file != NULL) {
        fwrite(extension.c_str(), 1, extension.size() + 1, file);  // +1 to include null terminator
        fclose(file);
    } else {
        std::cerr << "Failed to open file: " << filename << std::endl;
    }
}