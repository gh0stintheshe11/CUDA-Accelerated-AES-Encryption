#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

int main() {
    unsigned char *plaintext;  // Host plaintext
    unsigned char *ciphertext; // Host ciphertext
    unsigned char *d_plaintext, *d_ciphertext, *d_key;
    unsigned long long int *d_nonceCounter;
    int dataSize = 1024; // Example data size, adjust as needed
    unsigned char key[AES_KEY_SIZE]; // AESi key, ensure AES_KEY_SIZE is defined
    unsigned long long int nonceCounter = 0; // Example nonceCounter, initialize appropriately

    // Allocate host memory
    plaintext = (unsigned char*)malloc(dataSize * sizeof(unsigned char));
    ciphertext = (unsigned char*)malloc(dataSize * sizeof(unsigned char));

    // Initialize plaintext and key as needed

    // Allocate device memory
    hipMalloc((void **)&d_plaintext, dataSize * sizeof(unsigned char));
    hipMalloc((void **)&d_ciphertext, dataSize * sizeof(unsigned char));
    hipMalloc((void **)&d_key, AES_KEY_SIZE * sizeof(unsigned char));
    hipMalloc((void **)&d_nonceCounter, sizeof(unsigned long long int));

    // Copy host memory to device
    hipMemcpy(d_plaintext, plaintext, dataSize * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_key, key, AES_KEY_SIZE * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_nonceCounter, &nonceCounter, sizeof(unsigned long long int), hipMemcpyHostToDevice);

    // Define block and grid sizes
    int blockSize = 256; // Example, can be optimized
    int numBlocks = (dataSize + blockSize - 1) / blockSize;

    // Launch AES-CTR encryption kernel
    aes_ctr_encrypt_kernel<<<numBlocks, blockSize>>>(d_plaintext, d_ciphertext, d_key, d_nonceCounter, dataSize);

    // Copy device ciphertext back to host
    hipMemcpy(ciphertext, d_ciphertext, dataSize * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_plaintext);
    hipFree(d_ciphertext);
    hipFree(d_key);
    hipFree(d_nonceCounter);
    free(plaintext);
    free(ciphertext);

    return 0;
}
int main() {
    unsigned char *plaintext;  // Host plaintext
    unsigned char *ciphertext; // Host ciphertext
    unsigned char *d_plaintext, *d_ciphertext, *d_key;
    unsigned long long int *d_nonceCounter;
    int dataSize = 1024; // Example data size, adjust as needed
    unsigned char key[AES_KEY_SIZE]; // AESi key, ensure AES_KEY_SIZE is defined
    unsigned long long int nonceCounter = 0; // Example nonceCounter, initialize appropriately

    // Allocate host memory
    plaintext = (unsigned char*)malloc(dataSize * sizeof(unsigned char));
    ciphertext = (unsigned char*)malloc(dataSize * sizeof(unsigned char));

    // Initialize plaintext and key as needed

    // Allocate device memory
    hipMalloc((void **)&d_plaintext, dataSize * sizeof(unsigned char));
    hipMalloc((void **)&d_ciphertext, dataSize * sizeof(unsigned char));
    hipMalloc((void **)&d_key, AES_KEY_SIZE * sizeof(unsigned char));
    hipMalloc((void **)&d_nonceCounter, sizeof(unsigned long long int));

    // Copy host memory to device
    hipMemcpy(d_plaintext, plaintext, dataSize * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_key, key, AES_KEY_SIZE * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_nonceCounter, &nonceCounter, sizeof(unsigned long long int), hipMemcpyHostToDevice);

    // Define block and grid sizes
    int blockSize = 256; // Example, can be optimized
    int numBlocks = (dataSize + blockSize - 1) / blockSize;

    // Launch AES-CTR encryption kernel
    aes_ctr_encrypt_kernel<<<numBlocks, blockSize>>>(d_plaintext, d_ciphertext, d_key, d_nonceCounter, dataSize);

    // Copy device ciphertext back to host
    hipMemcpy(ciphertext, d_ciphertext, dataSize * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_plaintext);
    hipFree(d_ciphertext);
    hipFree(d_key);
    hipFree(d_nonceCounter);
    free(plaintext);
    free(ciphertext);

    return 0;
}
