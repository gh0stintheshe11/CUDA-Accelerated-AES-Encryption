#include "hip/hip_runtime.h"
#include "utils-cuda.h"
#include <chrono>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/*
    For benchmarking
    Optimization:
        -v1 Constant Memory: S box
        -v1 Shared Memory: IV and expanded key
        -v1 Pinned Memory: plaintext and ciphertext
*/

// Declare fixed data in constant memory
__constant__ unsigned char d_sbox_v1[256];

__global__ void aes_ctr_encrypt_kernel_v1(unsigned char *plaintext,
                                          unsigned char *ciphertext,
                                          unsigned char *expandedKey,
                                          unsigned char *iv, int numBlocks,
                                          int dataSize) {

  // Calculate the unique thread ID within the grid
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  // Create shared memory arrays for the IV and expanded key
  __shared__ unsigned char shared_iv[AES_BLOCK_SIZE];
  __shared__ unsigned char shared_expandedKey[176];

  // Copy the IV and expanded key to shared memory
  if (threadIdx.x < AES_BLOCK_SIZE) {
    shared_iv[threadIdx.x] = iv[threadIdx.x];
  }
  if (threadIdx.x < 176) {
    shared_expandedKey[threadIdx.x] = expandedKey[threadIdx.x];
  }

  // Synchronize to make sure the arrays are fully loaded
  __syncthreads();

  // Define the counter and initialize it with the IV
  unsigned char counter[AES_BLOCK_SIZE];
  memcpy(counter, shared_iv, AES_BLOCK_SIZE);

  // Increment the counter by the block ID
  increment_counter(counter, tid);

  // Calculate the block size
  int blockSize = (tid == numBlocks - 1 && dataSize % AES_BLOCK_SIZE != 0)
                      ? dataSize % AES_BLOCK_SIZE
                      : AES_BLOCK_SIZE;

  // Encrypt the counter to get the ciphertext block
  unsigned char ciphertextBlock[AES_BLOCK_SIZE];
  aes_encrypt_block(counter, ciphertextBlock, shared_expandedKey, d_sbox_v1);

  // XOR the plaintext with the ciphertext block
  for (int i = 0; i < blockSize; ++i) {
    ciphertext[tid * AES_BLOCK_SIZE + i] =
        plaintext[tid * AES_BLOCK_SIZE + i] ^ ciphertextBlock[i];
  }
}

std::pair<double, double> aes_encrypt_cuda_v1(unsigned char *plaintext,
                                              size_t dataSize,
                                              unsigned char *key,
                                              unsigned char *iv,
                                              unsigned char *ciphertext) {
  auto start = std::chrono::high_resolution_clock::now();

  unsigned char *d_plaintext, *d_ciphertext, *d_iv;
  unsigned char *d_expandedKey;

  // Call the host function to expand the key
  unsigned char expandedKey[176];
  KeyExpansionHost(key, expandedKey);

  // Calculate the number of AES blocks needed
  size_t numBlocks = (dataSize + AES_BLOCK_SIZE - 1) / AES_BLOCK_SIZE;

  // Define the size of the grid and the blocks
  dim3 threadsPerBlock(256); // Use a reasonable number of threads per block
  dim3 blocksPerGrid((numBlocks + threadsPerBlock.x - 1) / threadsPerBlock.x);

  // Allocate device memory
  hipMalloc((void **)&d_iv, AES_BLOCK_SIZE * sizeof(unsigned char));
  hipMalloc((void **)&d_expandedKey, 176);
  hipMalloc((void **)&d_plaintext, dataSize * sizeof(unsigned char));
  hipMalloc((void **)&d_ciphertext, dataSize * sizeof(unsigned char));

  // Copy S-box to device constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(d_sbox_v1), h_sbox, sizeof(h_sbox));

  // Copy host memory to device
  hipMemcpy(d_plaintext, plaintext, dataSize * sizeof(unsigned char),
             hipMemcpyHostToDevice);
  hipMemcpy(d_iv, iv, AES_BLOCK_SIZE * sizeof(unsigned char),
             hipMemcpyHostToDevice);
  hipMemcpy(d_expandedKey, expandedKey, 176, hipMemcpyHostToDevice);

  // Launch AES-CTR encryption kernel
  auto kernel_start = std::chrono::high_resolution_clock::now();
  aes_ctr_encrypt_kernel_v1<<<blocksPerGrid, threadsPerBlock>>>(
      d_plaintext, d_ciphertext, d_expandedKey, d_iv, numBlocks, dataSize);

  // Synchronize device
  hipDeviceSynchronize();
  auto kernel_stop = std::chrono::high_resolution_clock::now();

  // Copy device ciphertext back to host
  hipMemcpy(ciphertext, d_ciphertext, dataSize * sizeof(unsigned char),
             hipMemcpyDeviceToHost);

  // Get the stop time
  auto stop = std::chrono::high_resolution_clock::now();

  // Cleanup
  hipFree(d_plaintext);
  hipFree(d_ciphertext);
  hipFree(d_iv);
  hipFree(d_expandedKey);

  // Calculate the elapsed time and print
  return std::make_pair(
      std::chrono::duration_cast<std::chrono::microseconds>(stop - start)
          .count(),
      std::chrono::duration_cast<std::chrono::microseconds>(kernel_stop -
                                                            kernel_start)
          .count());
}