#include "hip/hip_runtime.h"
#include "bm-utils-cuda.h"
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <mutex>
#include <string>
#include <iostream>

unsigned char h_sbox[256] = {
    0x63, 0x7C, 0x77, 0x7B, 0xF2, 0x6B, 0x6F, 0xC5, 0x30, 0x01, 0x67, 0x2B, 0xFE, 0xD7, 0xAB, 0x76,
    0xCA, 0x82, 0xC9, 0x7D, 0xFA, 0x59, 0x47, 0xF0, 0xAD, 0xD4, 0xA2, 0xAF, 0x9C, 0xA4, 0x72, 0xC0,
    0xB7, 0xFD, 0x93, 0x26, 0x36, 0x3F, 0xF7, 0xCC, 0x34, 0xA5, 0xE5, 0xF1, 0x71, 0xD8, 0x31, 0x15,
    0x04, 0xC7, 0x23, 0xC3, 0x18, 0x96, 0x05, 0x9A, 0x07, 0x12, 0x80, 0xE2, 0xEB, 0x27, 0xB2, 0x75,
    0x09, 0x83, 0x2C, 0x1A, 0x1B, 0x6E, 0x5A, 0xA0, 0x52, 0x3B, 0xD6, 0xB3, 0x29, 0xE3, 0x2F, 0x84,
    0x53, 0xD1, 0x00, 0xED, 0x20, 0xFC, 0xB1, 0x5B, 0x6A, 0xCB, 0xBE, 0x39, 0x4A, 0x4C, 0x58, 0xCF,
    0xD0, 0xEF, 0xAA, 0xFB, 0x43, 0x4D, 0x33, 0x85, 0x45, 0xF9, 0x02, 0x7F, 0x50, 0x3C, 0x9F, 0xA8,
    0x51, 0xA3, 0x40, 0x8F, 0x92, 0x9D, 0x38, 0xF5, 0xBC, 0xB6, 0xDA, 0x21, 0x10, 0xFF, 0xF3, 0xD2,
    0xCD, 0x0C, 0x13, 0xEC, 0x5F, 0x97, 0x44, 0x17, 0xC4, 0xA7, 0x7E, 0x3D, 0x64, 0x5D, 0x19, 0x73,
    0x60, 0x81, 0x4F, 0xDC, 0x22, 0x2A, 0x90, 0x88, 0x46, 0xEE, 0xB8, 0x14, 0xDE, 0x5E, 0x0B, 0xDB,
    0xE0, 0x32, 0x3A, 0x0A, 0x49, 0x06, 0x24, 0x5C, 0xC2, 0xD3, 0xAC, 0x62, 0x91, 0x95, 0xE4, 0x79,
    0xE7, 0xC8, 0x37, 0x6D, 0x8D, 0xD5, 0x4E, 0xA9, 0x6C, 0x56, 0xF4, 0xEA, 0x65, 0x7A, 0xAE, 0x08,
    0xBA, 0x78, 0x25, 0x2E, 0x1C, 0xA6, 0xB4, 0xC6, 0xE8, 0xDD, 0x74, 0x1F, 0x4B, 0xBD, 0x8B, 0x8A,
    0x70, 0x3E, 0xB5, 0x66, 0x48, 0x03, 0xF6, 0x0E, 0x61, 0x35, 0x57, 0xB9, 0x86, 0xC1, 0x1D, 0x9E,
    0xE1, 0xF8, 0x98, 0x11, 0x69, 0xD9, 0x8E, 0x94, 0x9B, 0x1E, 0x87, 0xE9, 0xCE, 0x55, 0x28, 0xDF,
    0x8C, 0xA1, 0x89, 0x0D, 0xBF, 0xE6, 0x42, 0x68, 0x41, 0x99, 0x2D, 0x0F, 0xB0, 0x54, 0xBB, 0x16
};

unsigned char h_rcon[11] = {
    0x00, // not used
    0x01, 0x02, 0x04, 0x08, 0x10, 
    0x20, 0x40, 0x80, 0x1B, 0x36
};

// Function to read key or IV from a file
void read_key_or_iv(unsigned char *data, size_t size, const char *filename) {
    FILE *file = fopen(filename, "r");
    if (file == NULL) {
        fprintf(stderr, "Cannot open file: %s\n", filename);
        exit(1);
    }
    for (size_t i = 0; i < size; i++) {
        char buffer[3];
        if (fread(buffer, 1, 2, file) != 2) {
            fprintf(stderr, "Cannot read value from file: %s\n", filename);
            exit(1);
        }
        buffer[2] = '\0'; // Null-terminate the buffer
        data[i] = (unsigned char)strtol(buffer, NULL, 16); // Convert the buffer to a hexadecimal value
    }
    fclose(file);
}

void read_file_as_binary(unsigned char **data, size_t *size, const char *filename) {
    FILE *file = fopen(filename, "rb");
    if (file == NULL) {
        fprintf(stderr, "Cannot open file: %s\n", filename);
        exit(1);
    }

    // Determine the file size
    fseek(file, 0, SEEK_END);
    *size = ftell(file);
    fseek(file, 0, SEEK_SET);

    // Allocate the buffer
    *data = new unsigned char[*size];

    size_t bytesRead = fread(*data, 1, *size, file);
    if (bytesRead != *size) {
        fprintf(stderr, "Failed to read the entire file: %s\n", filename);
        exit(1);
    }

    fclose(file);
}

// Add pinned memory allocation
void read_file_as_binary_v2(unsigned char **data, size_t *size, const char *filename) {
    FILE *file = fopen(filename, "rb");
    if (file == NULL) {
        fprintf(stderr, "Cannot open file: %s\n", filename);
        exit(1);
    }

    // Determine the file size
    fseek(file, 0, SEEK_END);
    *size = ftell(file);
    fseek(file, 0, SEEK_SET);

    // Allocate pinned memory for data
    hipHostMalloc((void**)data, *size * sizeof(unsigned char));

    // Read the file into data
    if (fread(*data, 1, *size, file) != *size) {
        fprintf(stderr, "Cannot read file: %s\n", filename);
        exit(1);
    }

    fclose(file);
}

size_t preprocess(const char *filename, size_t chunkSize, unsigned char ***chunks, size_t **chunkSizes) {
    // Read the file into a buffer
    unsigned char *buffer;
    size_t bufferSize;
    read_file_as_binary_v2(&buffer, &bufferSize, filename);

    // Calculate the number of chunks
    size_t numChunks = (bufferSize + chunkSize - 1) / chunkSize;

    // Allocate pinned memory for the chunks and their sizes
    hipHostMalloc((void***)chunks, numChunks * sizeof(unsigned char*));
    hipHostMalloc((void**)chunkSizes, numChunks * sizeof(size_t));

    // Split the buffer into chunks
    for (size_t i = 0; i < numChunks; i++) {
        // Calculate the size of the current chunk
        size_t currentChunkSize = (i < numChunks - 1) ? chunkSize : (bufferSize % chunkSize);

        // Allocate pinned memory for the current chunk
        hipHostMalloc((void**)&(*chunks)[i], currentChunkSize * sizeof(unsigned char));

        printf("Chunk %zu address: %p\n", i, (*chunks)[i]);  // Print the address of the current chunk

        // Copy the data from the buffer to the current chunk
        memcpy((*chunks)[i], buffer + i * chunkSize, currentChunkSize);

        // Save the size of the current chunk
        (*chunkSizes)[i] = currentChunkSize;
    }

    // Free the buffer
    hipHostFree(buffer);

    return numChunks;
}

void write_encrypted(const unsigned char *ciphertext, size_t size, const char *filename) {
    FILE *file = fopen(filename, "wb");
    if (file == NULL) {
        fprintf(stderr, "Cannot open file: %s\n", filename);
        exit(1);
    }
    if (fwrite(ciphertext, 1, size, file) != size) {
        fprintf(stderr, "Error writing to file: %s\n", filename);
        exit(1);
    }
    fclose(file);
}

void write_encrypted_v2(unsigned char* ciphertext, size_t size, const char* filename) {
    FILE* file = fopen(filename, "ab");
    if (file == NULL) {
        printf("Error opening file: %s\n", filename);
        return;
    }
    if (fwrite(ciphertext, 1, size, file) != size) {
        fprintf(stderr, "Error writing to file: %s\n", filename);
        exit(1);
    }
    fclose(file);
}

std::mutex fileMutex;
void write_encrypted_multithreading(const unsigned char *ciphertext, size_t size, const char *filename) {
    std::lock_guard<std::mutex> lock(fileMutex);

    // Open the file in append mode
    FILE *file = fopen(filename, "ab");
    if (file == NULL) {
        fprintf(stderr, "Cannot open file: %s\n", filename);
        exit(1);
    }

    // Write the data to the file
    size_t written = fwrite(ciphertext, 1, size, file);
    if (written != size) {
        fprintf(stderr, "Failed to write to file: %s\n", filename);
        exit(1);
    }

    fclose(file);
}

std::string getFileExtension(const std::string& filename) {
    size_t pos = filename.rfind('.');
    return (pos == std::string::npos) ? "" : filename.substr(pos);
}

void appendFileExtension(const std::string& filename, const std::string& extension) {
    FILE* file = fopen(filename.c_str(), "ab");
    if (file != NULL) {
        fwrite(extension.c_str(), 1, extension.size() + 1, file);  // +1 to include null terminator
        fclose(file);
    } else {
        std::cerr << "Failed to open file: " << filename << std::endl;
    }
}

__device__ unsigned char mul(unsigned char a, unsigned char b) {
  unsigned char p = 0;
  unsigned char high_bit_mask = 0x80;
  unsigned char high_bit = 0;
  unsigned char modulo = 0x1B; /* x^8 + x^4 + x^3 + x + 1 */

  for (int i = 0; i < 8; i++) {
    if (b & 1) {
      p ^= a;
    }

    high_bit = a & high_bit_mask;
    a <<= 1;
    if (high_bit) {
      a ^= modulo;
    }
    b >>= 1;
  }

  return p;
}

void KeyExpansionHost(unsigned char *key, unsigned char *expandedKey) {
  int i = 0;
  while (i < 4) {
    for (int j = 0; j < 4; j++) {
      expandedKey[i * 4 + j] = key[i * 4 + j];
    }
    i++;
  }

  int rconIteration = 1;
  unsigned char temp[4];

  while (i < 44) {
    for (int j = 0; j < 4; j++) {
      temp[j] = expandedKey[(i - 1) * 4 + j];
    }

    if (i % 4 == 0) {
      unsigned char k = temp[0];
      for (int j = 0; j < 3; j++) {
        temp[j] = temp[j + 1];
      }
      temp[3] = k;

      for (int j = 0; j < 4; j++) {
        // Use the host-accessible arrays
        temp[j] = h_sbox[temp[j]] ^ (j == 0 ? h_rcon[rconIteration++] : 0);
      }
    }

    for (int j = 0; j < 4; j++) {
      expandedKey[i * 4 + j] = expandedKey[(i - 4) * 4 + j] ^ temp[j];
    }
    i++;
  }
}

__device__ void SubBytes(unsigned char *state, unsigned char *d_sbox) {
  for (int i = 0; i < 16; ++i) {
    state[i] = d_sbox[state[i]];
  }
}

__device__ void ShiftRows(unsigned char *state) {
  unsigned char tmp[16];

  /* Column 1 */
  tmp[0] = state[0];
  tmp[1] = state[5];
  tmp[2] = state[10];
  tmp[3] = state[15];
  /* Column 2 */
  tmp[4] = state[4];
  tmp[5] = state[9];
  tmp[6] = state[14];
  tmp[7] = state[3];
  /* Column 3 */
  tmp[8] = state[8];
  tmp[9] = state[13];
  tmp[10] = state[2];
  tmp[11] = state[7];
  /* Column 4 */
  tmp[12] = state[12];
  tmp[13] = state[1];
  tmp[14] = state[6];
  tmp[15] = state[11];

  memcpy(state, tmp, 16);
}

__device__ void MixColumns(unsigned char *state) {
  unsigned char tmp[16];

  for (int i = 0; i < 4; ++i) {
    tmp[i * 4] =
        (unsigned char)(mul(0x02, state[i * 4]) ^ mul(0x03, state[i * 4 + 1]) ^
                        state[i * 4 + 2] ^ state[i * 4 + 3]);
    tmp[i * 4 + 1] =
        (unsigned char)(state[i * 4] ^ mul(0x02, state[i * 4 + 1]) ^
                        mul(0x03, state[i * 4 + 2]) ^ state[i * 4 + 3]);
    tmp[i * 4 + 2] = (unsigned char)(state[i * 4] ^ state[i * 4 + 1] ^
                                     mul(0x02, state[i * 4 + 2]) ^
                                     mul(0x03, state[i * 4 + 3]));
    tmp[i * 4 + 3] =
        (unsigned char)(mul(0x03, state[i * 4]) ^ state[i * 4 + 1] ^
                        state[i * 4 + 2] ^ mul(0x02, state[i * 4 + 3]));
  }

  memcpy(state, tmp, 16);
}

__device__ void AddRoundKey(unsigned char *state,
                            const unsigned char *roundKey) {
  for (int i = 0; i < 16; ++i) {
    state[i] ^= roundKey[i];
  }
}

__device__ void aes_encrypt_block(unsigned char *input, unsigned char *output,
                                  unsigned char *expandedKey,
                                  unsigned char *d_sbox) {
  unsigned char state[16];

  // Copy the input to the state array
  for (int i = 0; i < 16; ++i) {
    state[i] = input[i];
  }

  // Add the round key to the state
  AddRoundKey(state, expandedKey);

  // Perform 9 rounds of substitutions, shifts, mixes, and round key additions
  for (int round = 1; round < 10; ++round) {
    SubBytes(state, d_sbox);
    ShiftRows(state);
    MixColumns(state);
    AddRoundKey(state, expandedKey + round * 16);
  }

  // Perform the final round (without MixColumns)
  SubBytes(state, d_sbox);
  ShiftRows(state);
  AddRoundKey(state, expandedKey + 10 * 16);

  // Copy the state to the output
  for (int i = 0; i < 16; ++i) {
    output[i] = state[i];
  }
}

__device__ void increment_counter(unsigned char *counter, int increment) {
  int carry = increment;
  for (int i = AES_BLOCK_SIZE - 1; i >= 0; i--) {
    int sum = counter[i] + carry;
    counter[i] = sum & 0xFF;
    carry = sum >> 8;
    if (carry == 0) {
      break;
    }
  }
}

__device__ unsigned char mul_v2(unsigned char a, unsigned char b) {
  unsigned char p = 0;
  unsigned char high_bit_mask = 0x80;
  unsigned char high_bit = 0;
  unsigned char modulo = 0x1B; /* x^8 + x^4 + x^3 + x + 1 */

  for (int i = 0; i < 8; i++) {
    p ^= a * (b & 1); // Use arithmetic instead of conditional

    high_bit = a & high_bit_mask;
    a <<= 1;
    a ^= modulo * (high_bit >> 7); // Use arithmetic instead of conditional
    b >>= 1;
  }

  return p;
}

__device__ void MixColumns_v2(unsigned char *state) {
  unsigned char tmp[16];

  for (int i = 0; i < 4; ++i) {
    tmp[i * 4] = (unsigned char)(mul_v2(0x02, state[i * 4]) ^
                                 mul_v2(0x03, state[i * 4 + 1]) ^
                                 state[i * 4 + 2] ^ state[i * 4 + 3]);
    tmp[i * 4 + 1] =
        (unsigned char)(state[i * 4] ^ mul_v2(0x02, state[i * 4 + 1]) ^
                        mul_v2(0x03, state[i * 4 + 2]) ^ state[i * 4 + 3]);
    tmp[i * 4 + 2] = (unsigned char)(state[i * 4] ^ state[i * 4 + 1] ^
                                     mul_v2(0x02, state[i * 4 + 2]) ^
                                     mul_v2(0x03, state[i * 4 + 3]));
    tmp[i * 4 + 3] =
        (unsigned char)(mul_v2(0x03, state[i * 4]) ^ state[i * 4 + 1] ^
                        state[i * 4 + 2] ^ mul_v2(0x02, state[i * 4 + 3]));
  }

  memcpy(state, tmp, 16);
}

__device__ void aes_encrypt_block_v2(unsigned char *input,
                                     unsigned char *output,
                                     unsigned char *expandedKey,
                                     unsigned char *d_sbox) {
  unsigned char state[16];

  // Copy the input to the state array
  for (int i = 0; i < 16; ++i) {
    state[i] = input[i];
  }

  // Add the round key to the state
  AddRoundKey(state, expandedKey);

  // Perform 9 rounds of substitutions, shifts, mixes, and round key additions
  for (int round = 1; round < 10; ++round) {
    SubBytes(state, d_sbox);
    ShiftRows(state);
    MixColumns_v2(state);
    AddRoundKey(state, expandedKey + round * 16);
  }

  // Perform the final round (without MixColumns_v2)
  SubBytes(state, d_sbox);
  ShiftRows(state);
  AddRoundKey(state, expandedKey + 10 * 16);

  // Copy the state to the output
  for (int i = 0; i < 16; ++i) {
    output[i] = state[i];
  }
}