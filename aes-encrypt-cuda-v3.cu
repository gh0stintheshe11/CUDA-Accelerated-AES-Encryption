#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include "utils-cuda.h"

/*
    Memory optimizations:
        - Add shared memory for data with in SM -> slightly improve kernel throughput
        - Add constant memory for expanded key and IV -> slightly improve kernel throughput
        - Add stream for GPU kernel -> transfer data still waste time

    Kernel optimization:
        - CUDA intrinsic function (fast build-in functions): use __byte_perm in the ShiftRow(), use __mul24 in the mul(), __shfl_sync in kernel()
        - 
*/

#define AES_KEY_SIZE 16
#define AES_BLOCK_SIZE 16

__constant__ unsigned char d_sbox[256];
__constant__ unsigned char d_rcon[11];
// Declare constant memory variables for the IV and expanded key
__constant__ unsigned char constantIv[AES_BLOCK_SIZE];
__constant__ unsigned char constantExpandedKey[176];

// Host function to copy the IV and expanded key to constant memory
void copyToConstantMemory(unsigned char *iv, unsigned char *expandedKey) {
    hipMemcpyToSymbol(HIP_SYMBOL(constantIv), iv, AES_BLOCK_SIZE);
    hipMemcpyToSymbol(HIP_SYMBOL(constantExpandedKey), expandedKey, 176);
}

__device__ unsigned char mul(unsigned char a, unsigned char b) {
    unsigned char p = 0;
    unsigned char high_bit_mask = 0x80;
    unsigned char high_bit = 0;
    unsigned char modulo = 0x1B; /* x^8 + x^4 + x^3 + x + 1 */

    for (int i = 0; i < 8; i++) {
        if (b & 1) {
            p ^= a;
        }

        high_bit = a & high_bit_mask;
        a = __byte_perm(a, 0, 0x1011); // shift left
        if (high_bit) {
            a ^= modulo;
        }
        b >>= 1;
    }

    return p;
}

void KeyExpansionHost(unsigned char* key, unsigned char* expandedKey) {
    int i = 0;
    while (i < 4) {
        hipMemcpy(&expandedKey[i * 4], &key[i * 4], 4 * sizeof(unsigned char), hipMemcpyHostToHost);
        i++;
    }

    int rconIteration = 1;
    unsigned char temp[4];

    while (i < 44) {
        hipMemcpy(temp, &expandedKey[(i - 1) * 4], 4 * sizeof(unsigned char), hipMemcpyHostToHost);

        if (i % 4 == 0) {
            unsigned char k = temp[0];
            for (int j = 0; j < 3; j++) {
                temp[j] = temp[j + 1];
            }
            temp[3] = k;

            for (int j = 0; j < 4; j++) {
                // Use the host-accessible arrays
                temp[j] = h_sbox[temp[j]] ^ (j == 0 ? h_rcon[rconIteration++] : 0);
            }
        }

        for (int j = 0; j < 4; j++) {
            expandedKey[i * 4 + j] = expandedKey[(i - 4) * 4 + j] ^ temp[j];
        }
        i++;
    }
}

__device__ void SubBytes(unsigned char *state) {
    for (int i = 0; i < 16; ++i) {
        state[i] = d_sbox[state[i]];
    }
}

__device__ void ShiftRows(unsigned char *state) {
    uint4 *state_as_int4 = reinterpret_cast<uint4*>(state);
    uint4 state0 = state_as_int4[0];
    uint4 state1 = state_as_int4[1];
    uint4 state2 = state_as_int4[2];
    uint4 state3 = state_as_int4[3];

    state_as_int4[0] = make_uint4(__byte_perm(state0.x, state1.x, 0x3210),
                                  __byte_perm(state0.y, state1.y, 0x3210),
                                  __byte_perm(state0.z, state1.z, 0x3210),
                                  __byte_perm(state0.w, state1.w, 0x3210));

    state_as_int4[1] = make_uint4(__byte_perm(state1.x, state2.x, 0x3210),
                                  __byte_perm(state1.y, state2.y, 0x3210),
                                  __byte_perm(state1.z, state2.z, 0x3210),
                                  __byte_perm(state1.w, state2.w, 0x3210));

    state_as_int4[2] = make_uint4(__byte_perm(state2.x, state3.x, 0x3210),
                                  __byte_perm(state2.y, state3.y, 0x3210),
                                  __byte_perm(state2.z, state3.z, 0x3210),
                                  __byte_perm(state2.w, state3.w, 0x3210));

    state_as_int4[3] = make_uint4(__byte_perm(state3.x, state0.x, 0x3210),
                                  __byte_perm(state3.y, state0.y, 0x3210),
                                  __byte_perm(state3.z, state0.z, 0x3210),
                                  __byte_perm(state3.w, state0.w, 0x3210));
}

__device__ void MixColumns(unsigned char *state) {
    unsigned char tmp[16];

    for (int i = 0; i < 4; ++i) {
        tmp[i*4] = (unsigned char)(mul(0x02, state[i*4]) ^ mul(0x03, state[i*4+1]) ^ state[i*4+2] ^ state[i*4+3]);
        tmp[i*4+1] = (unsigned char)(state[i*4] ^ mul(0x02, state[i*4+1]) ^ mul(0x03, state[i*4+2]) ^ state[i*4+3]);
        tmp[i*4+2] = (unsigned char)(state[i*4] ^ state[i*4+1] ^ mul(0x02, state[i*4+2]) ^ mul(0x03, state[i*4+3]));
        tmp[i*4+3] = (unsigned char)(mul(0x03, state[i*4]) ^ state[i*4+1] ^ state[i*4+2] ^ mul(0x02, state[i*4+3]));
    }

    memcpy(state, tmp, 16);
}

__device__ void AddRoundKey(unsigned char *state, const unsigned char *roundKey) {
    for (int i = 0; i < 16; ++i) {
        state[i] ^= roundKey[i];
    }
}

__device__ void aes_encrypt_block(unsigned char *input, unsigned char *output, unsigned char *expandedKey) {
    unsigned char state[16];

    // Copy the input to the state array
    #pragma unroll
    for (int i = 0; i < 16; ++i) {
        state[i] = input[i];
    }

    // Add the round key to the state
    AddRoundKey(state, expandedKey);

    // Perform 9 rounds of substitutions, shifts, mixes, and round key additions
    for (int round = 1; round < 10; ++round) {
        SubBytes(state);
        ShiftRows(state);
        MixColumns(state);
        AddRoundKey(state, expandedKey + round * 16);
    }

    // Perform the final round (without MixColumns)
    SubBytes(state);
    ShiftRows(state);
    AddRoundKey(state, expandedKey + 10 * 16);

    // Copy the state to the output
    #pragma unroll
    for (int i = 0; i < 16; ++i) {
        output[i] = state[i];
    }
}

__global__ void aes_ctr_encrypt_kernel(unsigned char *plaintext, unsigned char *ciphertext, int numBlocks) {
    // Calculate the global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Declare shared memory for the IV
    __shared__ unsigned char sharedIv[AES_BLOCK_SIZE];

    // Load the IV into shared memory
    if (threadIdx.x < AES_BLOCK_SIZE) {
        sharedIv[threadIdx.x] = constantIv[threadIdx.x];
    }

    // Synchronize to make sure the data is loaded before proceeding
    __syncthreads();

    // Check if the thread is within the number of blocks
    if (tid < numBlocks) {
        // Copy the IV to a local array
        unsigned char localIv[AES_BLOCK_SIZE];
        memcpy(localIv, sharedIv, AES_BLOCK_SIZE);

        // Increment the counter in the local IV
        for (int i = AES_BLOCK_SIZE - 1; i >= 0; --i) {
            if (++localIv[i] != 0) break;  // Increment the current byte and break if there's no carry
        }

        // Perform the AES encryption
        unsigned char block[AES_BLOCK_SIZE];
        aes_encrypt_block(localIv, block, constantExpandedKey);  // Use constantExpandedKey here

        // XOR the plaintext with the encrypted block
        // __shfl_sync is used to exchange the block array between the threads in a warp. The 0xffffffff mask indicates that all threads in the warp participate in the shuffle operation. The block[i] is the value to be shuffled, and threadIdx.x is the source lane.
        #pragma unroll
        for (int i = 0; i < AES_BLOCK_SIZE; ++i) {
            unsigned char block_i = __shfl_sync(0xffffffff, block[i], threadIdx.x);
            ciphertext[tid * AES_BLOCK_SIZE + i] = plaintext[tid * AES_BLOCK_SIZE + i] ^ block_i;
        }
    }
}

int main() {

    // Read the key and IV
    unsigned char key[16];
    unsigned char iv[16];
    read_key_or_iv(key, sizeof(key), "key.txt");
    read_key_or_iv(iv, sizeof(iv), "iv.txt");

    // Determine the size of the file and read the plaintext
    size_t dataSize;
    unsigned char *plaintext;
    read_file_as_binary(&plaintext, &dataSize, "plaintext.txt"); 

    unsigned char *d_plaintext, *d_ciphertext;

    // Copy S-box and rcon to device constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(d_sbox), h_sbox, sizeof(h_sbox));
    hipMemcpyToSymbol(HIP_SYMBOL(d_rcon), h_rcon, sizeof(h_rcon));

    // Call the host function to expand the key
    unsigned char expandedKey[176];
    KeyExpansionHost(key, expandedKey);

    // Copy the IV and expanded key to constant memory
    copyToConstantMemory(iv, expandedKey);

    // Calculate the number of AES blocks needed
    size_t numBlocks = (dataSize + AES_BLOCK_SIZE - 1) / AES_BLOCK_SIZE;

    // Define the size of the grid and the blocks
    dim3 threadsPerBlock(256); // Use a reasonable number of threads per block
    dim3 blocksPerGrid((numBlocks + threadsPerBlock.x - 1) / threadsPerBlock.x);

    // Allocate device memory
    hipMalloc((void **)&d_plaintext, numBlocks * AES_BLOCK_SIZE * sizeof(unsigned char));
    hipMalloc((void **)&d_ciphertext, numBlocks * AES_BLOCK_SIZE * sizeof(unsigned char));

    // Allocate memory for the ciphertext on the host
    unsigned char *ciphertext = new unsigned char[dataSize];

    // Copy host memory to device
    hipMemcpy(d_plaintext, plaintext, dataSize * sizeof(unsigned char), hipMemcpyHostToDevice);

    // Set the rest of d_plaintext to zero
    hipMemset(d_plaintext + dataSize, 0, numBlocks * AES_BLOCK_SIZE - dataSize);

    // Determine the number of streams based on the number of SMs
    int numStreams = 16;  // Use full 82 will decrese performance, best at 8 and 16

    // Create the streams
    hipStream_t *streams = new hipStream_t[numStreams];
    for (int i = 0; i < numStreams; ++i) {
        hipStreamCreate(&streams[i]);
    }

    // Calculate the number of blocks per stream
    size_t blocksPerStream = (numBlocks + numStreams - 1) / numStreams;

    // Loop over the streams
    for (int i = 0; i < numStreams; ++i) {
        // Calculate the start and end block for this stream
        size_t startBlock = i * blocksPerStream;
        size_t endBlock = min(startBlock + blocksPerStream, numBlocks);

        // Check if there are any blocks for this stream
        if (startBlock < endBlock) {
            // Calculate the number of blocks and threads for this stream
            dim3 blocks(endBlock - startBlock);
            dim3 threads(AES_BLOCK_SIZE);

            // Launch the kernel in this stream
            aes_ctr_encrypt_kernel<<<blocks, threads, 0, streams[i]>>>(d_plaintext + startBlock * AES_BLOCK_SIZE, d_ciphertext + startBlock * AES_BLOCK_SIZE, endBlock - startBlock);
        }
    }

    // Wait for all streams to finish
    for (int i = 0; i < numStreams; ++i) {
        hipStreamSynchronize(streams[i]);
    }

    // Clean up
    for (int i = 0; i < numStreams; ++i) {
        hipStreamDestroy(streams[i]);
    }
    delete[] streams;

    // Copy device ciphertext back to host
    hipMemcpy(ciphertext, d_ciphertext, dataSize * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Output encoded text to a file
    write_ciphertext(ciphertext, dataSize, "ciphertext.bin");

    // Cleanup
    hipFree(d_plaintext);
    hipFree(d_ciphertext);
    delete[] ciphertext;
    delete[] plaintext; 
    return 0;
}
