#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <string.h>
#include <vector>
#include <thread>
#include <queue>
#include <mutex>
#include <chrono>
#include <condition_variable>
#include "utils-cuda.h"

/*
    DEAD END, do not use.
    CPU multithreading (MT) + GPU stream version
    I can't get this goddamn piece of shit working!
    also maybe too slow: guessing a big transfer is better than multiple small transfer?
*/

#define AES_KEY_SIZE 16
#define AES_BLOCK_SIZE 16

__constant__ unsigned char d_sbox[256];
__constant__ unsigned char d_rcon[11];

__device__ unsigned char mul(unsigned char a, unsigned char b) {
    unsigned char p = 0;
    unsigned char high_bit_mask = 0x80;
    unsigned char high_bit = 0;
    unsigned char modulo = 0x1B; /* x^8 + x^4 + x^3 + x + 1 */

    for (int i = 0; i < 8; i++) {
        if (b & 1) {
            p ^= a;
        }

        high_bit = a & high_bit_mask;
        a <<= 1;
        if (high_bit) {
            a ^= modulo;
        }
        b >>= 1;
    }

    return p;
}

void KeyExpansionHost(unsigned char* key, unsigned char* expandedKey) {
    int i = 0;
    while (i < 4) {
        for (int j = 0; j < 4; j++) {
            expandedKey[i * 4 + j] = key[i * 4 + j];
        }
        i++;
    }

    int rconIteration = 1;
    unsigned char temp[4];

    while (i < 44) {
        for (int j = 0; j < 4; j++) {
            temp[j] = expandedKey[(i - 1) * 4 + j];
        }

        if (i % 4 == 0) {
            unsigned char k = temp[0];
            for (int j = 0; j < 3; j++) {
                temp[j] = temp[j + 1];
            }
            temp[3] = k;

            for (int j = 0; j < 4; j++) {
                // Use the host-accessible arrays
                temp[j] = h_sbox[temp[j]] ^ (j == 0 ? h_rcon[rconIteration++] : 0);
            }
        }

        for (int j = 0; j < 4; j++) {
            expandedKey[i * 4 + j] = expandedKey[(i - 4) * 4 + j] ^ temp[j];
        }
        i++;
    }
}

__device__ void SubBytes(unsigned char *state) {
    for (int i = 0; i < 16; ++i) {
        state[i] = d_sbox[state[i]];
    }
}

__device__ void ShiftRows(unsigned char *state) {
    unsigned char tmp[16];

    /* Column 1 */
    tmp[0] = state[0];
    tmp[1] = state[5];
    tmp[2] = state[10];
    tmp[3] = state[15];
    /* Column 2 */
    tmp[4] = state[4];
    tmp[5] = state[9];
    tmp[6] = state[14];
    tmp[7] = state[3];
    /* Column 3 */
    tmp[8] = state[8];
    tmp[9] = state[13];
    tmp[10] = state[2];
    tmp[11] = state[7];
    /* Column 4 */
    tmp[12] = state[12];
    tmp[13] = state[1];
    tmp[14] = state[6];
    tmp[15] = state[11];

    memcpy(state, tmp, 16);
}

__device__ void MixColumns(unsigned char *state) {
    unsigned char tmp[16];

    for (int i = 0; i < 4; ++i) {
        tmp[i*4] = (unsigned char)(mul(0x02, state[i*4]) ^ mul(0x03, state[i*4+1]) ^ state[i*4+2] ^ state[i*4+3]);
        tmp[i*4+1] = (unsigned char)(state[i*4] ^ mul(0x02, state[i*4+1]) ^ mul(0x03, state[i*4+2]) ^ state[i*4+3]);
        tmp[i*4+2] = (unsigned char)(state[i*4] ^ state[i*4+1] ^ mul(0x02, state[i*4+2]) ^ mul(0x03, state[i*4+3]));
        tmp[i*4+3] = (unsigned char)(mul(0x03, state[i*4]) ^ state[i*4+1] ^ state[i*4+2] ^ mul(0x02, state[i*4+3]));
    }

    memcpy(state, tmp, 16);
}

__device__ void AddRoundKey(unsigned char *state, const unsigned char *roundKey) {
    for (int i = 0; i < 16; ++i) {
        state[i] ^= roundKey[i];
    }
}

__device__ void aes_encrypt_block(unsigned char *input, unsigned char *output, unsigned char *expandedKey) {
    unsigned char state[16];

    // Copy the input to the state array
    for (int i = 0; i < 16; ++i) {
        state[i] = input[i];
    }

    // Add the round key to the state
    AddRoundKey(state, expandedKey);

    // Perform 9 rounds of substitutions, shifts, mixes, and round key additions
    for (int round = 1; round < 10; ++round) {
        SubBytes(state);
        ShiftRows(state);
        MixColumns(state);
        AddRoundKey(state, expandedKey + round * 16);
    }

    // Perform the final round (without MixColumns)
    SubBytes(state);
    ShiftRows(state);
    AddRoundKey(state, expandedKey + 10 * 16);

    // Copy the state to the output
    for (int i = 0; i < 16; ++i) {
        output[i] = state[i];
    }
}

__device__ void increment_counter(unsigned char *counter, int increment) {
    int carry = increment;
    for (int i = AES_BLOCK_SIZE - 1; i >= 0; i--) {
        int sum = counter[i] + carry;
        counter[i] = sum & 0xFF;
        carry = sum >> 8;
        if (carry == 0) {
            break;
        }
    }
}

__global__ void aes_ctr_encrypt_kernel(unsigned char *plaintext, unsigned char *ciphertext, unsigned char *expandedKey, unsigned char *iv, int numBlocks, int dataSize) {
    // Calculate the global block ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the block is within the number of blocks
    if (tid < numBlocks) {
        // Create a counter array
        unsigned char counter[AES_BLOCK_SIZE];

        // Copy the IV to the counter
        memcpy(counter, iv, AES_BLOCK_SIZE);

        // Increment the counter by the block ID
        increment_counter(counter, tid);

        // Calculate the block size
        int blockSize = (tid == numBlocks - 1 && dataSize % AES_BLOCK_SIZE != 0) ? dataSize % AES_BLOCK_SIZE : AES_BLOCK_SIZE;

        // Encrypt the counter to get the ciphertext block
        unsigned char ciphertextBlock[AES_BLOCK_SIZE];
        aes_encrypt_block(counter, ciphertextBlock, expandedKey);

        // XOR the plaintext with the ciphertext block
        for (int i = 0; i < blockSize; ++i) {
            ciphertext[tid * AES_BLOCK_SIZE + i] = plaintext[tid * AES_BLOCK_SIZE + i] ^ ciphertextBlock[i];
        }
    }
}

std::queue<size_t> workQueue;
std::mutex queueMutex;
std::condition_variable queueCondVar;

bool allChunksProcessed = false;

unsigned char** chunks;
size_t numChunks;
size_t* chunkSizes;

struct Chunk {
    size_t number;
    unsigned char* data;
    size_t size;
};

// Create a std::deque to hold the chunks to be written to the file
std::deque<Chunk> writeQueue;
std::mutex writeMutex;
std::condition_variable writeCondVar;

void writeThread(const char* filename) {
    while (true) {
        Chunk chunk;

        {
            std::unique_lock<std::mutex> lock(writeMutex);
            writeCondVar.wait(lock, []{ return !writeQueue.empty(); });

            if (!writeQueue.empty()) {
                // Get the next processed chunk from the queue
                chunk = writeQueue.front();
                writeQueue.pop_front();
                std::cout << "WriteThread: Got chunk " << chunk.number << std::endl;

                // Write the processed chunk to the output file
                write_encrypted_multithreading(chunk.data, chunk.size, filename);
                std::cout << "WriteThread: Writing chunk " << chunk.number << " to file " << filename << std::endl;
            }

            if (writeQueue.empty() && allChunksProcessed) {
                std::cout << "WriteThread: All chunks processed, exiting" << std::endl;
                break;
            }
        }
    }
}

void processChunk(size_t i, unsigned char** chunks, unsigned char** ciphertexts, size_t* chunkSizes, unsigned char* expandedKey, unsigned char* iv, hipStream_t* streams, unsigned char** d_chunks, unsigned char** d_ciphertexts) {
    std::cout << "Processing chunk " << i << std::endl;
    hipStreamCreate(&streams[i]);

    // Allocate memory on the GPU
    hipMalloc(&d_chunks[i], chunkSizes[i]);
    hipMalloc(&d_ciphertexts[i], chunkSizes[i]);

    // Allocate memory for iv on the GPU
    unsigned char* d_iv;
    hipMalloc(&d_iv, AES_BLOCK_SIZE);
    hipMemcpy(d_iv, iv, AES_BLOCK_SIZE, hipMemcpyHostToDevice);

    // Copy the chunk to the GPU
    hipMemcpyAsync(d_chunks[i], chunks[i], chunkSizes[i], hipMemcpyHostToDevice, streams[i]);

    // Launch the kernel
    dim3 numThreadsPerBlock(256);
    dim3 numBlocksPerGrid((chunkSizes[i] + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
    aes_ctr_encrypt_kernel<<<numBlocksPerGrid, numThreadsPerBlock>>>(d_chunks[i], d_ciphertexts[i], expandedKey, d_iv, numBlocksPerGrid.x, chunkSizes[i]);

    // Copy the processed data back to the CPU
    hipMemcpyAsync(ciphertexts[i], d_ciphertexts[i], chunkSizes[i], hipMemcpyDeviceToHost, streams[i]);

    // Wait for the copy to finish
    hipStreamSynchronize(streams[i]);

    hipFree(d_chunks[i]);
    hipFree(d_ciphertexts[i]);
    hipFree(d_iv);
    hipStreamDestroy(streams[i]);

    // After the copy is finished, add the chunk to the writeQueue
    {
        std::lock_guard<std::mutex> lock(writeMutex);
        writeQueue.push_back({i, ciphertexts[i], chunkSizes[i]});
        writeCondVar.notify_one();
    }
    std::cout << "Finished processing chunk " << i << std::endl;
}

void workerThread(unsigned char** chunks, unsigned char** ciphertexts, size_t* chunkSizes, unsigned char expandedKey[176], unsigned char iv[16], hipStream_t* streams, unsigned char** d_chunks, unsigned char** d_ciphertexts) {
    while (true) {
        size_t i;
        // Get a chunk from the workQueue
        {
            std::unique_lock<std::mutex> lock(queueMutex);
            while (workQueue.empty()) {
                if (allChunksProcessed) {
                    return;
                }
                queueCondVar.wait(lock);
            }
            // Get the next chunk number from the queue
            i = workQueue.front();
            workQueue.pop();
        }
        // Process the chunk
        processChunk(i, chunks, ciphertexts, chunkSizes, expandedKey, iv, streams, d_chunks, d_ciphertexts);

        // After the chunk is processed, add it to the writeQueue
        {
            std::lock_guard<std::mutex> lock(writeMutex);
            unsigned char* ciphertextCopy = new unsigned char[chunkSizes[i]];
            memcpy(ciphertextCopy, ciphertexts[i], chunkSizes[i]);
            writeQueue.push_back({i, ciphertextCopy, chunkSizes[i]});
            writeCondVar.notify_one();
        }

        // Delete the original ciphertexts[i] after it's copied and added to the writeQueue
        delete[] ciphertexts[i];
    }
}

int main(int argc, char* argv[]) {
    // Check if filename is provided
    if (argc < 2) {
        printf("Usage: %s <filename>\n", argv[0]);
        return 1;
    }

    // Get the start time
    auto start = std::chrono::high_resolution_clock::now();

    // Read the key and IV
    unsigned char key[16];
    unsigned char iv[16];
    read_key_or_iv(key, sizeof(key), "key.txt");
    read_key_or_iv(iv, sizeof(iv), "iv.txt");

    unsigned char** ciphertexts = new unsigned char*[numChunks];
    for (size_t i = 0; i < numChunks; i++) {
        ciphertexts[i] = new unsigned char[chunkSizes[i]];
    }

    // Call the host function to expand the key
    unsigned char expandedKey[176];
    KeyExpansionHost(key, expandedKey);

    // Preprocess the data into chunks
    numChunks = preprocess(argv[1], AES_BLOCK_SIZE, &chunks, &chunkSizes);

    // Create a pool of CUDA streams
    hipStream_t* streams = new hipStream_t[numChunks];
    unsigned char** d_chunks = new unsigned char*[numChunks];
    unsigned char** d_ciphertexts = new unsigned char*[numChunks];

    // Create the worker threads
    std::thread workerThreads[8];
    for (int i = 0; i < 8; i++) {
        workerThreads[i] = std::thread(workerThread, chunks, ciphertexts, chunkSizes, expandedKey, iv, streams, d_chunks, d_ciphertexts);
    }

    // Create the write thread
    std::thread writeThreadInstance(writeThread, "encrypted.bin");

    // Add the chunks to the work queue
    for (size_t i = 0; i < numChunks; i++) {
        {
            std::lock_guard<std::mutex> lock(queueMutex);
            workQueue.push(i);
        }
        queueCondVar.notify_one();
        std::cout << "Added chunk " << i << " to work queue" << std::endl;
    }

    // Notify all waiting threads that all chunks have been processed
    queueCondVar.notify_all();

    // Wait for all threads to finish
    for (int i = 0; i < 8; i++) {
        workerThreads[i].join();
    }

    // Set allChunksProcessed to true after all chunks have been processed
    allChunksProcessed = true;
    std::cout << "WorkerThread: All chunks processed, exiting" << std::endl;

    writeCondVar.notify_all();

    // Wait for the write thread to finish
    writeThreadInstance.join();

    delete[] chunks;
    delete[] chunkSizes;
    delete[] streams;
    delete[] d_chunks;
    delete[] d_ciphertexts;

    // Get the stop time
    auto stop = std::chrono::high_resolution_clock::now();

    // Calculate the elapsed time and print
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
    std::cout << "Elapsed time: " << duration.count() << " ms\n";

    return 0;
}